//===- op_registry.cu --------------------------------------------- C++ ---===//
//
// Copyright 2023 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "flux/op_registry_proto_utils.h"
#include "flux/flux.h"
#include "flux/op_registry.h"
#include <mutex>

namespace bytedance {
namespace flux {

namespace {
std::once_flag init_flag;
ArchEnum arch;

void
init_arch_tag() {
  int major, minor;
  hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0);
  hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);
  int arch_num = major * 10 + minor;
  FLUX_CHECK(arch_num == 80 || arch_num == 89 || arch_num == 90)
      << "unsupported arch: " << arch_num;
  arch = ArchEnum{arch_num};
}
}  // namespace

ArchEnum
get_arch() {
  std::call_once(init_flag, init_arch_tag);
  return arch;
}

TuningConfigRegistry &
TuningConfigRegistry::instance() {
  static TuningConfigRegistry inst;
  char *env = getenv("FLUX_TUNE_CONFIG_FILE");
  if (env != nullptr) {
    static std::once_flag flag;
    std::call_once(flag, load_tune_config_from_file, inst, std::string(env));
  } else {
#if defined(FLUX_DEBUG)
    std::cerr
        << "FLUX_TUNE_CONFIG_FILE not set. no tune config file specified, using default configs\n";
#endif
  }
  return inst;
}

OpRegistry &
OpRegistry::instance() {
  static OpRegistry inst;
  return inst;
}

}  // namespace flux
}  // namespace bytedance
