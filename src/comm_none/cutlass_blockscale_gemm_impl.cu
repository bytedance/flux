//===- cutlass_blockscale_gemm_impl.cu ---------------------------- C++ ---===//
//
// Copyright 2025 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "comm_none/cutlass_blockscale_gemm_impl.h"
#include "flux/args/comm_none.h"
#include "flux/cuda/cuda_common.h"

namespace bytedance {
namespace flux {
namespace {
using namespace cute;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
struct CutlassBlockScaleGemmImpl {
  // A matrix configuration
  using ElementA = cutlass::float_e4m3_t;     // Element type for A matrix operand
  using LayoutA = cutlass::layout::RowMajor;  // Layout type for A matrix operand
  static constexpr int AlignmentA =
      128 / cutlass::sizeof_bits<ElementA>::value;  // Memory access granularity/alignment of A
                                                    // matrix in units of elements (up to 16 bytes)

  // B matrix configuration
  using ElementB = cutlass::float_e4m3_t;        // Element type for B matrix operand
  using LayoutB = cutlass::layout::ColumnMajor;  // Layout type for B matrix operand
  static constexpr int AlignmentB =
      128 / cutlass::sizeof_bits<ElementB>::value;  // Memory access granularity/alignment of B
                                                    // matrix in units of elements (up to 16 bytes)

  // C matrix configuration
  using ElementC = void;                          // Element type for C and D matrix operands
  using ElementCNonVoid = cutlass::float_e4m3_t;  // Element type for C and D matrix operands
  using LayoutC = cutlass::layout::ColumnMajor;   // Layout type for C and D matrix operands
  static constexpr int AlignmentC =
      128 / cutlass::sizeof_bits<ElementCNonVoid>::value;  // Memory access granularity/alignment
                                                           // of C matrix in units of elements (up
                                                           // to 16 bytes)

  // D matrix configuration
  using ElementD = cutlass::float_e4m3_t;
  using LayoutD = cutlass::layout::RowMajor;
  static constexpr int AlignmentD = AlignmentC;

  // Auxiliary matrix configuration and other fusion types
  using ElementAux = ElementCNonVoid;
  using LayoutAux = LayoutC;
  using ElementAmax = float;
  using ElementBias = float;

  // Core kernel configurations
  using ElementAccumulator = float;  // Element type for internal accumulation
  using ElementBlockScale = float;   // Element type for blockscaling during accumulation
  using ElementCompute = float;      // Element type for epilogue computation
  using ArchTag =
      cutlass::arch::Sm90;  // Tag indicating the minimum SM that supports the intended feature
  using OperatorClass = cutlass::arch::OpClassTensorOp;  // Operator class tag
  using TileShape = Shape<_128, _128, _128>;             // Threadblock-level tile size
  using ClusterShape = Shape<_1, _2, _1>;                // Shape of the threadblocks in a cluster

  static constexpr int ScaleMsPerTile = 128;
  static constexpr int ScaleGranularityM = size<0>(TileShape{}) / ScaleMsPerTile;

  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedCooperativeFP8BlockScaledAccum<ScaleGranularityM>;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;

  using EpilogueTileType = cutlass::epilogue::collective::EpilogueTileAuto;
  using FusionOperation = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltActAmaxAux<
      LayoutAux,
      cutlass::epilogue::thread::Identity,
      ElementD,
      ElementCompute,
      ElementAux,
      ElementAmax,
      ElementBias,
      ElementCNonVoid>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      TileShape,
      ClusterShape,
      EpilogueTileType,
      ElementAccumulator,
      ElementCompute,
      ElementC,
      LayoutC,
      AlignmentC,
      ElementD,
      LayoutD,
      AlignmentD,
      EpilogueSchedule,
      FusionOperation>::CollectiveOp;

  using CollectiveMainloopWithBlockWiseScaling =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          ElementA,
          LayoutA,
          AlignmentA,
          ElementB,
          LayoutB,
          AlignmentB,
          ElementAccumulator,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          KernelSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,  // Indicates ProblemShape
      CollectiveMainloopWithBlockWiseScaling,
      CollectiveEpilogue>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  static typename Gemm::Arguments
  args_from_flux(const BlockScaleGemmArguments &args) {
    auto ptr_A = static_cast<ElementA const *>(args.A);
    auto ptr_B = static_cast<ElementB const *>(args.B);
    auto ptr_C = static_cast<ElementCNonVoid const *>(args.C);
    auto ptr_D = static_cast<ElementD *>(args.D);
    auto ptr_blockscale_A = static_cast<ElementBlockScale const *>(args.blockscale_A);
    auto ptr_blockscale_B = static_cast<ElementBlockScale const *>(args.blockscale_B);

    auto stride_A = cutlass::make_cute_packed_stride(
        typename GemmKernel::StrideA{}, cute::make_shape(args.m, args.k, args.l));
    auto stride_B = cutlass::make_cute_packed_stride(
        typename GemmKernel::StrideB{}, cute::make_shape(args.n, args.k, args.l));
    auto stride_C = cutlass::make_cute_packed_stride(
        typename GemmKernel::StrideC{}, cute::make_shape(args.m, args.n, args.l));
    auto stride_D = cutlass::make_cute_packed_stride(
        typename GemmKernel::StrideD{}, cute::make_shape(args.m, args.n, args.l));

    typename Gemm::Arguments arguments{
        cutlass::gemm::GemmUniversalMode::kGemm,
        {args.m, args.n, args.k, args.l},
        {ptr_A,
         stride_A,
         ptr_B,
         stride_B,
         args.mma_promotion_interval,
         ptr_blockscale_A,
         ptr_blockscale_B},
        {{},  // epilogue.thread
         ptr_C,
         stride_C,
         ptr_D,
         stride_D}};

    auto &fusion_args = arguments.epilogue.thread;
    fusion_args.alpha = args.alpha;
    fusion_args.beta = args.beta;
    fusion_args.alpha_ptr = nullptr;
    fusion_args.beta_ptr = nullptr;
    fusion_args.scale_a = args.scale_a;
    fusion_args.scale_b = args.scale_b;
    fusion_args.scale_c = args.scale_c;
    fusion_args.scale_a_ptr = nullptr;
    fusion_args.scale_b_ptr = nullptr;
    fusion_args.scale_c_ptr = nullptr;

    // ignored if tensor types are not fp8
    fusion_args.scale_d = args.scale_d;
    fusion_args.scale_aux = args.scale_aux;
    fusion_args.scale_d_ptr = nullptr;
    fusion_args.scale_aux_ptr = nullptr;

    // leaving/setting these as nullptr disables the fusion at runtime
    fusion_args.bias_ptr = nullptr;

    fusion_args.aux_ptr = nullptr;
    fusion_args.amax_aux_ptr = nullptr;
    fusion_args.amax_D_ptr = nullptr;

    return arguments;
  }

  static void
  run(const BlockScaleGemmArguments &flux_args, void *workspace, hipStream_t stream) {
    Gemm gemm;
    auto arguments = args_from_flux(flux_args);
    // Check if the problem size is supported or not
    CUTLASS_CHECK(gemm.can_implement(arguments));

    // Initialize CUTLASS kernel with arguments and workspace pointer
    CUTLASS_CHECK(gemm.initialize(arguments, workspace));

    // Correctness / Warmup iteration
    CUTLASS_CHECK(gemm.run());
  }

  static size_t
  get_workspace_size(const BlockScaleGemmArguments &flux_args) {
    auto arguments = args_from_flux(flux_args);
    size_t workspace_size = Gemm::get_workspace_size(arguments);
    return workspace_size;
  }
};
#endif
}  // namespace

void
CutlassBlockScaleGemm::run(
    const BlockScaleGemmArguments &flux_args, void *workspace, hipStream_t stream) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  CutlassBlockScaleGemmImpl::run(flux_args, workspace, stream);
#else
  assert(0);
#endif
}

size_t
CutlassBlockScaleGemm::get_workspace_size(const BlockScaleGemmArguments &flux_args) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  return CutlassBlockScaleGemmImpl::get_workspace_size(flux_args);
#else
  assert(0);
  return 0;
#endif
}

}  // namespace flux
}  // namespace bytedance
