#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define DEFAULT_ITERS 10
#define DEFAULT_MIN_MSG_SIZE 1
#define DEFAULT_MAX_MSG_SIZE 128 * 1024 * 1024

typedef enum { PUSH = 0, PULL = 1 } dir_t;

__global__ void test_kernel(void *data_d_local, long long int ncycles) {
    long long int sclk = clock64();
    long long int cyc = 0;
    while (cyc < ncycles) {
        cyc = clock64() - sclk;
    }
    *(long long int *)data_d_local = cyc;
}

int lat(void *data_d, void *data_d_local, int sizeBytes, int pe, int iter, dir_t dir,
        hipStream_t strm, hipEvent_t sev, hipEvent_t eev, float *ms1, float *ms2, int ng, int nb,
        long long int ncycles) {
    int status = 0;
    int peer = !pe;

    if (dir == PUSH) {
        CUDA_CHECK(hipEventRecord(sev, strm));
        for (int i = 0; i < iter; i++) {
            test_kernel<<<ng, nb, 0, strm>>>(data_d_local, ncycles);
            nvshmemx_putmem_on_stream((void *)data_d, (void *)data_d_local, sizeBytes, peer, strm);
        }
        CUDA_CHECK(hipEventRecord(eev, strm));
        CUDA_CHECK(hipEventSynchronize(eev));
        CUDA_CHECK(hipEventElapsedTime(ms1, sev, eev));

        CUDA_CHECK(hipEventRecord(sev, strm));
        for (int i = 0; i < iter; i++) {
            test_kernel<<<ng, nb, 0, strm>>>(data_d_local, ncycles);
            CUDA_CHECK(hipStreamSynchronize(strm));
            nvshmem_putmem((void *)data_d, (void *)data_d_local, sizeBytes, peer);
        }
        CUDA_CHECK(hipEventRecord(eev, strm));
        CUDA_CHECK(hipEventSynchronize(eev));
        CUDA_CHECK(hipEventElapsedTime(ms2, sev, eev));
    } else {
        CUDA_CHECK(hipEventRecord(sev, strm));
        for (int i = 0; i < iter; i++) {
            nvshmemx_getmem_on_stream((void *)data_d_local, (void *)data_d, sizeBytes, peer, strm);
            test_kernel<<<ng, nb, 0, strm>>>(data_d_local, ncycles);
        }
        CUDA_CHECK(hipEventRecord(eev, strm));
        CUDA_CHECK(hipEventSynchronize(eev));
        CUDA_CHECK(hipEventElapsedTime(ms1, sev, eev));

        CUDA_CHECK(hipEventRecord(sev, strm));
        for (int i = 0; i < iter; i++) {
            nvshmem_getmem((void *)data_d_local, (void *)data_d, sizeBytes,
                           peer);  // shmem_getmem is blocking, so nvshmem_quiet is not needed
            test_kernel<<<ng, nb, 0, strm>>>(data_d_local, ncycles);
        }
        CUDA_CHECK(hipEventRecord(eev, strm));
        CUDA_CHECK(hipEventSynchronize(eev));
        CUDA_CHECK(hipEventElapsedTime(ms2, sev, eev));
    }

    return status;
}

int main(int argc, char *argv[]) {
    int status = 0;
    int mype, npes;
    char *data_d = NULL, *data_d_local = NULL;
    uint64_t *size_array = NULL;
    double *offs_latency_array = NULL;
    double *ons_latency_array = NULL;
    int num_entries;
    int i;

    dir_t dir = PUSH;
    int iter = DEFAULT_ITERS;
    int min_msg_size = DEFAULT_MIN_MSG_SIZE;
    int max_msg_size = DEFAULT_MAX_MSG_SIZE;

    int nb = 1, nt = 32;
    long long int ncycles = 1;

    init_wrapper(&argc, &argv);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        status = -1;
        goto finalize;
    }

    while (1) {
        int c;
        c = getopt(argc, argv, "s:S:n:i:d:b:t:c:h");
        if (c == -1) break;

        switch (c) {
            case 's':
                min_msg_size = strtol(optarg, NULL, 0);
                break;
            case 'S':
                max_msg_size = strtol(optarg, NULL, 0);
                break;
            case 'n':
                iter = strtol(optarg, NULL, 0);
                break;
            case 'd':
                dir = (dir_t)strtol(optarg, NULL, 0);
                break;
            case 'b':
                nb = strtol(optarg, NULL, 0);
                break;
            case 't':
                nt = strtol(optarg, NULL, 0);
                break;
            case 'c':
                ncycles = strtol(optarg, NULL, 0);
                break;
            default:
            case 'h':
                printf(
                    "-n [Iterations] -S [Max message size] -s [Min message size] -i [Put/Get issue type : ON_STREAM(0) otherwise 1] -d [Direction of copy : PUSH(0) or PULL(1)] -b [# blocks] \
                 -t [# threads] -c [# cycles to wait in the the kernel]\n");
                goto finalize;
        }
    }

    num_entries = floor(std::log2((float)max_msg_size)) - floor(std::log2((float)min_msg_size)) + 1;
    size_array = (uint64_t *)calloc(sizeof(uint64_t), num_entries);
    if (!size_array) {
        status = -1;
        goto finalize;
    }

    offs_latency_array = (double *)calloc(sizeof(double), num_entries);
    if (!offs_latency_array) {
        status = -1;
        goto finalize;
    }

    ons_latency_array = (double *)calloc(sizeof(double), num_entries);
    if (!ons_latency_array) {
        status = -1;
        goto finalize;
    }

    data_d = (char *)nvshmem_malloc(max_msg_size);
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));

    data_d_local = (char *)nvshmem_malloc(max_msg_size);
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));

    hipStream_t strm;
    CUDA_CHECK(hipStreamCreateWithFlags(&strm, hipStreamNonBlocking));

    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) {
        float ms1, ms2;
        hipEvent_t sev, eev;
        CUDA_CHECK(hipEventCreate(&sev));
        CUDA_CHECK(hipEventCreate(&eev));
        i = 0;
        for (int size = min_msg_size; size <= max_msg_size; size *= 2) {
            size_array[i] = size;
            lat(data_d, data_d_local, size, mype, iter, dir, strm, sev, eev, &ms1, &ms2, nb, nt,
                ncycles);
            ons_latency_array[i] = ms1 / iter * 1000;
            offs_latency_array[i] = ms2 / iter * 1000;
            i++;
        }

        print_table_v1("Stream_Latency", "with _on_stream", "size (Bytes)", "latency", "us", '-',
                       size_array, ons_latency_array, i);
        print_table_v1("Stream_Latency", "without _on_stream", "size (Bytes)", "latency", "us", '-',
                       size_array, offs_latency_array, i);

        CUDA_CHECK(hipEventDestroy(sev));
        CUDA_CHECK(hipEventDestroy(eev));

        nvshmem_barrier_all();

    } else {
        nvshmem_barrier_all();
    }

finalize:
    CUDA_CHECK(hipStreamDestroy(strm));

    if (data_d) nvshmem_free(data_d);
    if (size_array) free(size_array);
    if (ons_latency_array) free(ons_latency_array);
    if (offs_latency_array) free(offs_latency_array);

    if (data_d_local) nvshmem_free(data_d_local);

    finalize_wrapper();

    return status;
}
