#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include "coll_test.h"
#define LARGEST_DT double2

#ifdef MAX_ITERS
#undef MAX_ITERS
#endif
#define MAX_ITERS 50

#define CALL_RDXN(TG_PRE, TG, TYPENAME, TYPE, OP, THREAD_COMP, ELEM_COMP)                     \
    __global__ void test_##TYPENAME##_##OP##_reduce_kern##TG(                                 \
        nvshmem_team_t team, TYPE *dest, const TYPE *source, int nelems, int iter) {          \
        int i;                                                                                \
                                                                                              \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP) && (nelems < ELEM_COMP)) {             \
            for (i = 0; i < iter; i++) {                                                      \
                nvshmem##TG_PRE##_##TYPENAME##_##OP##_reduce##TG(team, dest, source, nelems); \
            }                                                                                 \
        }                                                                                     \
    }

#define CALL_RDXN_OPS_ALL_TG(TYPENAME, TYPE) \
    CALL_RDXN(x, _block, TYPENAME, TYPE, maxloc, INT_MAX, 2)

CALL_RDXN_OPS_ALL_TG(double2, double2)

#define SET_SIZE_ARR(TYPE, ELEM_COMP)                                \
    do {                                                             \
        j = 0;                                                       \
        for (num_elems = 1; num_elems < max_elems; num_elems *= 2) { \
            if (num_elems < ELEM_COMP) {                             \
                size_arr[j] = num_elems * sizeof(TYPE);              \
            } else {                                                 \
                size_arr[j] = 0;                                     \
            }                                                        \
            j++;                                                     \
        }                                                            \
    } while (0)

#define RUN_ITERS_OP(TYPENAME, TYPE, GROUP, OP, ELEM_COMP)                             \
    do {                                                                               \
        void *skip_arg_list[] = {&team, &dest, &source, &num_elems, &skip};            \
        void *time_arg_list[] = {&team, &dest, &source, &num_elems, &iter};            \
        float milliseconds;                                                            \
        hipEvent_t start, stop;                                                       \
        hipEventCreate(&start);                                                       \
        hipEventCreate(&stop);                                                        \
        SET_SIZE_ARR(TYPE, ELEM_COMP);                                                 \
                                                                                       \
        nvshmem_barrier_all();                                                         \
        j = 0;                                                                         \
        for (num_elems = 1; num_elems < ELEM_COMP; num_elems *= 2) {                   \
            status = nvshmemx_collective_launch(                                       \
                (const void *)test_##TYPENAME##_##OP##_reduce_kern##GROUP, num_blocks, \
                nvshm_test_num_tpb, skip_arg_list, 0, stream);                         \
            if (status != NVSHMEMX_SUCCESS) {                                          \
                fprintf(stderr, "shmemx_collective_launch failed %d \n", status);      \
                exit(-1);                                                              \
            }                                                                          \
            CUDA_CHECK(hipStreamSynchronize(stream));                                 \
            nvshmem_barrier_all();                                                     \
                                                                                       \
            hipEventRecord(start, stream);                                            \
            status = nvshmemx_collective_launch(                                       \
                (const void *)test_##TYPENAME##_##OP##_reduce_kern##GROUP, num_blocks, \
                nvshm_test_num_tpb, time_arg_list, 0, stream);                         \
            if (status != NVSHMEMX_SUCCESS) {                                          \
                fprintf(stderr, "shmemx_collective_launch failed %d \n", status);      \
                exit(-1);                                                              \
            }                                                                          \
            hipEventRecord(stop, stream);                                             \
            CUDA_CHECK(hipStreamSynchronize(stream));                                 \
                                                                                       \
            if (!mype) {                                                               \
                hipEventElapsedTime(&milliseconds, start, stop);                      \
                h_##OP##_lat[j] = (milliseconds * 1000.0) / (float)iter;               \
            }                                                                          \
            nvshmem_barrier_all();                                                     \
            j++;                                                                       \
        }                                                                              \
    } while (0)

#define RUN_ITERS(TYPENAME, TYPE, GROUP, ELEM_COMP) \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, maxloc, ELEM_COMP);

int rdxn_calling_kernel(nvshmem_team_t team, void *dest, const void *source, int mype,
                        int max_elems, hipStream_t stream, run_opt_t run_options,
                        void **h_tables) {
    int status = 0;
    int nvshm_test_num_tpb = TEST_NUM_TPB_BLOCK;
    int num_blocks = 1;
    int num_elems = 1;
    int iter = MAX_ITERS;
    int skip = MAX_SKIP;
    int j;
    uint64_t *size_arr = (uint64_t *)h_tables[0];
    double *h_maxloc_lat = (double *)h_tables[1];

    if (run_options.run_block) {
        RUN_ITERS(double2, double2, _block, max_elems);
        if (!mype) {
            print_table_v1("device_reduction", "double2-maxloc-b", "size (Bytes)", "latency", "us",
                           '-', size_arr, h_maxloc_lat, j);
        }
    }

    return status;
}

int main(int argc, char **argv) {
    int status = 0;
    int mype, array_size;
    size_t size = 0;
    size_t alloc_size;
    int num_elems;
    char *value = NULL;
    int max_elems = 2;  //(MAX_ELEMS / 2);
    int *h_buffer = NULL;
    int *d_source, *d_dest;
    int *h_source, *h_dest;
    char size_string[100];
    hipStream_t cstrm;
    run_opt_t run_options;
    void **h_tables;

    PROCESS_OPTS(run_options);

    size = page_size_roundoff((MAX_ELEMS) * sizeof(LARGEST_DT));   // send buf
    size += page_size_roundoff((MAX_ELEMS) * sizeof(LARGEST_DT));  // recv buf

    DEBUG_PRINT("symmetric size requested %lu\n", size);
    sprintf(size_string, "%lu", size);

    status = setenv("NVSHMEM_SYMMETRIC_SIZE", size_string, 1);
    if (status) {
        fprintf(stderr, "setenv failed \n");
        status = -1;
        goto out;
    }

    value = getenv("NVSHMEM_PERF_COLL_MAX_ELEMS");

    if (NULL != value) {
        max_elems = atoi(value);
        if (0 == max_elems) {
            fprintf(stderr, "Warning: min max elem size = 1\n");
            max_elems = 1;
        }
    }

    array_size = floor(std::log2((float)max_elems)) + 1;

    init_wrapper(&argc, &argv);
    alloc_tables(&h_tables, 8, array_size);

    mype = nvshmem_my_pe();

    CUDA_CHECK(hipStreamCreateWithFlags(&cstrm, hipStreamNonBlocking));

    num_elems = 1;  // MAX_ELEMS / 2;
    alloc_size = (num_elems * 2) * sizeof(LARGEST_DT);

    CUDA_CHECK(hipHostAlloc(&h_buffer, alloc_size, hipHostMallocDefault));
    h_source = (int32_t *)h_buffer;
    h_dest = (int32_t *)&h_source[num_elems];

    d_source = (int32_t *)nvshmem_align(getpagesize(), num_elems * sizeof(LARGEST_DT));
    d_dest = (int32_t *)nvshmem_align(getpagesize(), num_elems * sizeof(LARGEST_DT));

    CUDA_CHECK(hipMemcpyAsync(d_source, h_source, (sizeof(LARGEST_DT) * num_elems),
                               hipMemcpyHostToDevice, cstrm));
    CUDA_CHECK(hipMemcpyAsync(d_dest, h_dest, (sizeof(LARGEST_DT) * num_elems),
                               hipMemcpyHostToDevice, cstrm));

    rdxn_calling_kernel(NVSHMEM_TEAM_WORLD, d_dest, d_source, mype, max_elems, cstrm, run_options,
                        h_tables);

    DEBUG_PRINT("last error = %s\n", hipGetErrorString(hipGetLastError()));

    CUDA_CHECK(hipMemcpyAsync(h_source, d_source, (sizeof(LARGEST_DT) * num_elems),
                               hipMemcpyDeviceToHost, cstrm));
    CUDA_CHECK(hipMemcpyAsync(h_dest, d_dest, (sizeof(LARGEST_DT) * num_elems),
                               hipMemcpyDeviceToHost, cstrm));

    nvshmem_barrier_all();

    CUDA_CHECK(hipHostFree(h_buffer));
    nvshmem_free(d_source);
    nvshmem_free(d_dest);

    CUDA_CHECK(hipStreamDestroy(cstrm));

    finalize_wrapper();

out:
    return 0;
}
