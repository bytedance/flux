#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include "coll_test.h"
#define DATATYPE int64_t

#define CALL_BCAST(TYPENAME, TYPE, TG_PRE, THREADGROUP, THREAD_COMP, ELEM_COMP)                   \
    __global__ void test_##TYPENAME##_bcast_call_kern##THREADGROUP(                               \
        nvshmem_team_t team, TYPE *dest, const TYPE *source, int nelems, int PE_root, int mype,   \
        int iter) {                                                                               \
        int i;                                                                                    \
                                                                                                  \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP) && (nelems < ELEM_COMP)) {                 \
            for (i = 0; i < iter; i++) {                                                          \
                nvshmem##TG_PRE##_##TYPENAME##_broadcast##THREADGROUP(team, dest, source, nelems, \
                                                                      PE_root);                   \
            }                                                                                     \
        }                                                                                         \
    }

CALL_BCAST(int32, int32_t, , , 1, 512);
CALL_BCAST(int64, int64_t, , , 1, 512);
CALL_BCAST(int32, int32_t, x, _warp, warpSize, 4096);
CALL_BCAST(int64, int64_t, x, _warp, warpSize, 4096);
CALL_BCAST(int32, int32_t, x, _block, INT_MAX, INT_MAX);
CALL_BCAST(int64, int64_t, x, _block, INT_MAX, INT_MAX);

int broadcast_calling_kernel(nvshmem_team_t team, void *dest, const void *source, int mype,
                             int PE_root, int max_elems, hipStream_t stream, void **h_tables) {
    int status = 0;
    int nvshm_test_num_tpb = TEST_NUM_TPB_BLOCK;
    int num_blocks = 1;
    int num_elems = 1;
    int i;
    int skip = MAX_SKIP;
    int iter = MAX_ITERS;
    uint64_t *h_size_array = (uint64_t *)h_tables[0];
    double *h_thread_lat = (double *)h_tables[1];
    double *h_warp_lat = (double *)h_tables[2];
    double *h_block_lat = (double *)h_tables[3];
    float milliseconds;
    void *args_1[] = {&team, &dest, &source, &num_elems, &mype, &PE_root, &skip};
    void *args_2[] = {&team, &dest, &source, &num_elems, &mype, &PE_root, &iter};
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float *ms_d = (float *)nvshmem_malloc(sizeof(float));
    float *ms_sum_d = (float *)nvshmem_malloc(sizeof(float));

    nvshmem_barrier_all();
    i = 0;
    for (num_elems = 1; num_elems < 512; num_elems *= 2) {
        status = nvshmemx_collective_launch((const void *)test_int32_bcast_call_kern, num_blocks,
                                            nvshm_test_num_tpb, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)test_int32_bcast_call_kern, num_blocks,
                                            nvshm_test_num_tpb, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_thread_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = 1; num_elems < 4096; num_elems *= 2) {
        status = nvshmemx_collective_launch((const void *)test_int32_bcast_call_kern_warp,
                                            num_blocks, nvshm_test_num_tpb, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)test_int32_bcast_call_kern_warp,
                                            num_blocks, nvshm_test_num_tpb, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_warp_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = 1; num_elems < max_elems; num_elems *= 2) {
        h_size_array[i] = num_elems * 4;
        status = nvshmemx_collective_launch((const void *)test_int32_bcast_call_kern_block,
                                            num_blocks, nvshm_test_num_tpb, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)test_int32_bcast_call_kern_block,
                                            num_blocks, nvshm_test_num_tpb, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_block_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    if (!mype) {
        print_table_v1("bcast_device", "32-bit-thread", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_thread_lat, i);
        print_table_v1("bcast_device", "32-bit-warp", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_warp_lat, i);
        print_table_v1("bcast_device", "32-bit-block", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_block_lat, i);
    }

    i = 0;
    for (num_elems = 1; num_elems < 512; num_elems *= 2) {
        status = nvshmemx_collective_launch((const void *)test_int64_bcast_call_kern, num_blocks,
                                            nvshm_test_num_tpb, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)test_int64_bcast_call_kern, num_blocks,
                                            nvshm_test_num_tpb, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_thread_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = 1; num_elems < 4096; num_elems *= 2) {
        status = nvshmemx_collective_launch((const void *)test_int64_bcast_call_kern_warp,
                                            num_blocks, nvshm_test_num_tpb, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)test_int64_bcast_call_kern_warp,
                                            num_blocks, nvshm_test_num_tpb, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_warp_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = 1; num_elems < max_elems; num_elems *= 2) {
        h_size_array[i] = num_elems * 8;
        status = nvshmemx_collective_launch((const void *)test_int64_bcast_call_kern_block,
                                            num_blocks, nvshm_test_num_tpb, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)test_int64_bcast_call_kern_block,
                                            num_blocks, nvshm_test_num_tpb, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        hipEventElapsedTime(&milliseconds, start, stop);
        hipMemcpy(ms_d, &milliseconds, sizeof(float), hipMemcpyHostToDevice);
        nvshmem_float_sum_reduce(NVSHMEM_TEAM_WORLD, ms_sum_d, ms_d, 1);
        hipMemcpy(&milliseconds, ms_sum_d, sizeof(float), hipMemcpyDeviceToHost);
        if (!mype) {
            h_block_lat[i] =
                (milliseconds * 1000.0) / ((float)iter * nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD));
        }
        i++;
        nvshmem_barrier_all();
    }

    if (!mype) {
        print_table_v1("bcast_device", "64-bit-thread", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_thread_lat, i);
        print_table_v1("bcast_device", "64-bit-warp", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_warp_lat, i);
        print_table_v1("bcast_device", "64-bit-block", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_block_lat, i);
    }

    return status;
}

int main(int argc, char **argv) {
    int status = 0;
    int mype, array_size, max_elems;
    char *value = NULL;
    size_t size = (MAX_ELEMS * 2) * sizeof(DATATYPE);
    size_t alloc_size;
    int num_elems;
    DATATYPE *buffer = NULL;
    DATATYPE *h_buffer = NULL;
    DATATYPE *d_source, *d_dest;
    DATATYPE *h_source, *h_dest;
    int root = 0;
    char size_string[100];
    hipStream_t cstrm;
    void **h_tables;

    max_elems = (MAX_ELEMS / 2);

    if (NULL != value) {
        max_elems = atoi(value);
        if (0 == max_elems) {
            fprintf(stderr, "Warning: min max elem size = 1\n");
            max_elems = 1;
        }
    }

    array_size = floor(std::log2((float)max_elems)) + 1;

    DEBUG_PRINT("symmetric size %lu\n", size);
    sprintf(size_string, "%lu", size);

    status = setenv("NVSHMEM_SYMMETRIC_SIZE", size_string, 1);
    if (status) {
        fprintf(stderr, "setenv failed \n");
        status = -1;
        goto out;
    }

    init_wrapper(&argc, &argv);
    alloc_tables(&h_tables, 4, array_size);

    mype = nvshmem_my_pe();
    CUDA_CHECK(hipStreamCreateWithFlags(&cstrm, hipStreamNonBlocking));

    DEBUG_PRINT("SHMEM: [%d of %d] hello shmem world! \n", mype,
                nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD));

    num_elems = MAX_ELEMS / 2;
    alloc_size = (num_elems * 2) * sizeof(DATATYPE);

    CUDA_CHECK(hipHostAlloc(&h_buffer, alloc_size, hipHostMallocDefault));
    h_source = (DATATYPE *)h_buffer;
    h_dest = (DATATYPE *)&h_source[num_elems];

    buffer = (DATATYPE *)nvshmem_malloc(alloc_size);
    if (!buffer) {
        fprintf(stderr, "nvshmem_malloc failed \n");
        status = -1;
        goto out;
    }

    d_source = (DATATYPE *)buffer;
    d_dest = (DATATYPE *)&d_source[num_elems];

    for (int i = 0; i < num_elems; i++) {
        h_source[i] = i;
    }

    CUDA_CHECK(hipMemcpyAsync(d_source, h_source, (sizeof(DATATYPE) * num_elems),
                               hipMemcpyHostToDevice, cstrm));
    CUDA_CHECK(hipMemcpyAsync(d_dest, h_dest, (sizeof(DATATYPE) * num_elems),
                               hipMemcpyHostToDevice, cstrm));

    broadcast_calling_kernel(NVSHMEM_TEAM_WORLD, d_dest, d_source, mype, root, max_elems, cstrm,
                             h_tables);

    CUDA_CHECK(hipMemcpyAsync(h_source, d_source, (sizeof(DATATYPE) * num_elems),
                               hipMemcpyDeviceToHost, cstrm));
    CUDA_CHECK(hipMemcpyAsync(h_dest, d_dest, (sizeof(DATATYPE) * num_elems),
                               hipMemcpyDeviceToHost, cstrm));

    nvshmem_barrier_all();

    CUDA_CHECK(hipHostFree(h_buffer));
    nvshmem_free(buffer);

    CUDA_CHECK(hipStreamDestroy(cstrm));
    free_tables(h_tables, 4);
    finalize_wrapper();

out:
    return 0;
}
