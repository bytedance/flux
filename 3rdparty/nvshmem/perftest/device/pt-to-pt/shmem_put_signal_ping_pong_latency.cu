#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define MAX_MSG_SIZE 1 * 1024 * 1024
#define UNROLL 8

#define PING_PONG(SC, SC_SUFFIX, SC_PREFIX)                                                   \
    __global__ void ping_pong##SC_SUFFIX(int *data_d, uint64_t *flag_d, int len, int pe,      \
                                         int iter) {                                          \
        int i, peer;                                                                          \
        int tid =                                                                             \
            (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z); \
        peer = !pe;                                                                           \
                                                                                              \
        for (i = 0; i < iter; i++) {                                                          \
            if (pe) {                                                                         \
                if (!tid) nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));         \
                nvshmem##SC_PREFIX##_int_put_signal_nbi##SC_SUFFIX(                           \
                    data_d, data_d, len, flag_d, i + 1, NVSHMEM_SIGNAL_SET, peer);            \
            } else {                                                                          \
                nvshmem##SC_PREFIX##_int_put_signal_nbi##SC_SUFFIX(                           \
                    data_d, data_d, len, flag_d, i + 1, NVSHMEM_SIGNAL_SET, peer);            \
                if (!tid) nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));         \
            }                                                                                 \
        }                                                                                     \
        nvshmem_quiet();                                                                      \
    }
PING_PONG(thread, , )
PING_PONG(warp, _warp, x)
PING_PONG(block, _block, x)

int main(int c, char *v[]) {
    int mype, npes, size;
    uint64_t *flag_d = NULL;
    int *data_d = NULL;
    hipStream_t stream;

    int iter = 500;
    int skip = 50;
    int max_msg_size = MAX_MSG_SIZE;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    float milliseconds;
    hipEvent_t start, stop;

    init_wrapper(&c, &v);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    data_d = (int *)nvshmem_malloc(max_msg_size);
    flag_d = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));
    CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));

    array_size = floor(std::log2((float)max_msg_size)) + 1;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    if (mype == 0) {
        printf("Note: This test measures full round-trip latency\n");
    }

    i = 0;
    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        int nelems, status = 0;
        nelems = size / sizeof(int);
        h_size_arr[i] = size;
        void *args_1[] = {&data_d, &flag_d, &nelems, &mype, &skip};
        void *args_2[] = {&data_d, &flag_d, &nelems, &mype, &iter};

        CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        CUDA_CHECK(hipDeviceSynchronize());
        nvshmem_barrier_all();

        status = nvshmemx_collective_launch((const void *)ping_pong, 1, 1, args_1, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        status = nvshmemx_collective_launch((const void *)ping_pong, 1, 1, args_2, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);

        /* give latency in us */
        CUDA_CHECK(hipEventSynchronize(stop));
        hipEventElapsedTime(&milliseconds, start, stop);
        h_lat[i] = (milliseconds * 1000) / iter;
        nvshmem_barrier_all();
        i++;
    }

    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) {
        print_table_v1("shmem_put_ping_lat", "None", "size (Bytes)", "latency", "us", '-',
                       h_size_arr, h_lat, i);
    }
finalize:

    if (data_d) nvshmem_free(data_d);
    if (flag_d) nvshmem_free(flag_d);
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
