/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure xor
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include "atomic_ping_pong_common.h"

/* add */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, add, (value * (1 + i)),
                                      (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int, int, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(long, long, add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(size_t, size, add, (value * (1 + i)), (value));

/* fetch_add */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, fetch_add, (value * (1 + i)),
                                      (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int, int, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(long, long, fetch_add, (value * (1 + i)), (value));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(size_t, size, fetch_add, (value * (1 + i)), (value));

/* and */
/* should get flag set to 0b1, 0b11, 0b111, etc. */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, and, (value << (i + 1)),
                                      (value << (i + 1)));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, and, (value << (i + 1)),
                                      (value << (i + 1)));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, and, (value << (i + 1)),
                                      (value << (i + 1)));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, and, (value << (i + 1)), (value << (i +
 * 1))); */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, and, (value << (i + 1)),
                                      (value << (i + 1)));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, and, (value << (i + 1)),
                                      (value << (i + 1)));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int64_t, int64, and, (value << (i + 1)), (value << (i +
 * 1))); */

/* fetch_and */
/* should get flag set to 0b1, 0b11, 0b111, etc. */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, fetch_and, (value << (i + 1)),
                                      (value << (i + 1)));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, fetch_and, (value << (i + 1)),
                                      (value << (i + 1)));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, fetch_and, (value << (i + 1)),
                                      (value << (i + 1)));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, fetch_and, (value << (i + 1)), (value << (i
 * + 1))); */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, fetch_and, (value << (i + 1)),
                                      (value << (i + 1)));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, fetch_and, (value << (i + 1)),
                                      (value << (i + 1)));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int64_t, int64, fetch_and, (value << (i + 1)), (value << (i
 * + 1))); */

/* inc */
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(unsigned int, uint, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(unsigned long, ulong, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(unsigned long long, ulonglong, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(int32_t, int32, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(uint32_t, uint32, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(uint64_t, uint64, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(int, int, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(long, long, inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(size_t, size, inc, (i + 1));

/* fetch_inc */
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(unsigned int, uint, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(unsigned long, ulong, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(unsigned long long, ulonglong, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(int32_t, int32, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(uint32_t, uint32, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(uint64_t, uint64, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(int, int, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(long, long, fetch_inc, (i + 1));
DEFINE_PING_PONG_TEST_FOR_AMO_NO_ARG(size_t, size, fetch_inc, (i + 1));

/* or */
/* should get flag set to 0b1, 0b11, 0b111, etc. */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, or, (cmp >> (iter - (i + 1))),
                                      (value << i));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, or, (cmp >> (iter - (i + 1))),
                                      (value << i));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, or, (cmp >> (iter - (i + 1))),
                                      (value << i));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, or, (cmp >> (iter - (i + 1))), (value <<
 * i)); */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, or, (cmp >> (iter - (i + 1))),
                                      (value << i));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, or, (cmp >> (iter - (i + 1))),
                                      (value << i));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int64_t, int64, or, (cmp >> (iter - (i + 1))), (value <<
 * i)); */

/* fetch_or */
/* should get flag set to 0b1, 0b11, 0b111, etc. */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, fetch_or, (cmp >> (iter - (i + 1))),
                                      (value << i));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, fetch_or, (cmp >> (iter - (i + 1))),
                                      (value << i));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, fetch_or,
                                      (cmp >> (iter - (i + 1))), (value << i));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, fetch_or, (cmp >> (iter - (i + 1))), (value
 * << i)); */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, fetch_or, (cmp >> (iter - (i + 1))),
                                      (value << i));
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, fetch_or, (cmp >> (iter - (i + 1))),
                                      (value << i));
/* DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int64_t, int64, fetch_or, (cmp >> (iter - (i + 1))), (value
 * << i)); */

/* xor */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int64_t, int64, xor, i % 2, 1);

/* fetch_xor */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, fetch_xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, fetch_xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, fetch_xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, fetch_xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, fetch_xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, fetch_xor, i % 2, 1);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int64_t, int64, fetch_xor, i % 2, 1);

/* set */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int, int, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(long, long, set, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(size_t, size, set, i, i);

/* swap */
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned int, uint, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long, ulong, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(unsigned long long, ulonglong, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int32_t, int32, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint32_t, uint32, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(uint64_t, uint64, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(int, int, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(long, long, swap, i, i);
DEFINE_PING_PONG_TEST_FOR_AMO_ONE_ARG(size_t, size, swap, i, i);

/* compare_swap */
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(unsigned int, uint, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(unsigned long, ulong, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(unsigned long long, ulonglong, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(int32_t, int32, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(uint32_t, uint32, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(uint64_t, uint64, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(int, int, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(long, long, compare_swap, i, i + 1);
DEFINE_PING_PONG_TEST_FOR_AMO_TWO_ARG(size_t, size, compare_swap, i, i + 1);

int main(int c, char *v[]) {
    int mype, npes;
    int iter, skip;
    int rc = 0;

    void *flag_d = NULL;
    hipStream_t stream;
    nvshmemi_amo_t op;

    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    MAIN_SETUP(c, v, mype, npes, flag_d, stream, h_size_arr, h_tables, h_lat, &op);

    switch (op) {
        case NVSHMEMI_AMO_INC: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITHOUT_ARG(unsigned int, uint, inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(unsigned long, ulong, inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(unsigned long long, ulonglong, inc, flag_d, mype, iter, skip,
                                 h_lat, h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(int32_t, int32, inc, flag_d, mype, iter, skip, h_lat, h_size_arr,
                                 0);
            RUN_TEST_WITHOUT_ARG(uint32_t, uint32, inc, flag_d, mype, iter, skip, h_lat, h_size_arr,
                                 0);
            RUN_TEST_WITHOUT_ARG(uint64_t, uint64, inc, flag_d, mype, iter, skip, h_lat, h_size_arr,
                                 0);
            RUN_TEST_WITHOUT_ARG(int, int, inc, flag_d, mype, iter, skip, h_lat, h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(long, long, inc, flag_d, mype, iter, skip, h_lat, h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(size_t, size, inc, flag_d, mype, iter, skip, h_lat, h_size_arr, 0);
            break;
        }
        case NVSHMEMI_AMO_SET: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned int, uint, set, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 551);
            RUN_TEST_WITH_ARG(unsigned long, ulong, set, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 551);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, set, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 551);
            RUN_TEST_WITH_ARG(int32_t, int32, set, flag_d, mype, iter, skip, h_lat, h_size_arr, 415,
                              0, 551);
            RUN_TEST_WITH_ARG(uint32_t, uint32, set, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 551);
            RUN_TEST_WITH_ARG(uint64_t, uint64, set, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 551);
            RUN_TEST_WITH_ARG(int, int, set, flag_d, mype, iter, skip, h_lat, h_size_arr, 415, 0,
                              551);
            RUN_TEST_WITH_ARG(long, long, set, flag_d, mype, iter, skip, h_lat, h_size_arr, 415, 0,
                              551);
            RUN_TEST_WITH_ARG(size_t, size, set, flag_d, mype, iter, skip, h_lat, h_size_arr, 415,
                              0, 551);
            break;
        }
        case NVSHMEMI_AMO_ADD: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned int, uint, add, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            RUN_TEST_WITH_ARG(unsigned long, ulong, add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(int32_t, int32, add, flag_d, mype, iter, skip, h_lat, h_size_arr, 415,
                              0, 0);
            RUN_TEST_WITH_ARG(uint32_t, uint32, add, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            RUN_TEST_WITH_ARG(uint64_t, uint64, add, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            RUN_TEST_WITH_ARG(int, int, add, flag_d, mype, iter, skip, h_lat, h_size_arr, 415, 0,
                              0);
            RUN_TEST_WITH_ARG(long, long, add, flag_d, mype, iter, skip, h_lat, h_size_arr, 415, 0,
                              0);
            RUN_TEST_WITH_ARG(size_t, size, add, flag_d, mype, iter, skip, h_lat, h_size_arr, 415,
                              0, 0);
            break;
        }
        case NVSHMEMI_AMO_AND: {
            iter = 64;
            skip = 0;
            /* TODO: Figure out a good way to do this with signed types. The bit shifts we do don't
             * mix with signed types. */
            RUN_TEST_WITH_ARG(unsigned long, ulong, and, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, and, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF);
            RUN_TEST_WITH_ARG(uint64_t, uint64, and, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF);
            /* RUN_TEST_WITH_ARG(int64_t, int64, and, flag_d, mype, iter, skip, h_lat, h_size_arr,
             * 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF); */
            iter = 32;
            /* RUN_TEST_WITH_ARG(int64_t, int64, and, flag_d, mype, iter, skip, h_lat, h_size_arr,
             * 0xFFFFFFFF, 0, 0xFFFFFFFF); */
            RUN_TEST_WITH_ARG(uint32_t, uint32, and, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0xFFFFFFFF, 0, 0xFFFFFFFF);
            RUN_TEST_WITH_ARG(unsigned int, uint, and, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0xFFFFFFFF, 0, 0xFFFFFFFF);
            break;
        }
        case NVSHMEMI_AMO_OR: {
            iter = 64;
            skip = 0;
            /* TODO: Figure out a good way to do this with signed types. The bit shifts we do don't
             * mix with signed types. */
            RUN_TEST_WITH_ARG(unsigned long, ulong, or, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              1, 0xFFFFFFFFFFFFFFFF, 0);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, or, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 1, 0xFFFFFFFFFFFFFFFF, 0);
            RUN_TEST_WITH_ARG(uint64_t, uint64, or, flag_d, mype, iter, skip, h_lat, h_size_arr, 1,
                              0xFFFFFFFFFFFFFFFF, 0);
            /* RUN_TEST_WITH_ARG(int64_t, int64, or, flag_d, mype, iter, skip, h_lat, h_size_arr, 1,
             * 0xFFFFFFFFFFFFFFFF, 0); */
            iter = 32;
            /* RUN_TEST_WITH_ARG(int64_t, int64, or, flag_d, mype, iter, skip, h_lat, h_size_arr, 1,
             * 0xFFFFFFFFFFFFFFFF, 0); */
            RUN_TEST_WITH_ARG(uint32_t, uint32, or, flag_d, mype, iter, skip, h_lat, h_size_arr, 1,
                              0xFFFFFFFF, 0);
            RUN_TEST_WITH_ARG(unsigned int, uint, or, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              1, 0xFFFFFFFF, 0);
            break;
        }
        case NVSHMEMI_AMO_XOR: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned long, ulong, xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(uint64_t, uint64, xor, flag_d, mype, iter, skip, h_lat, h_size_arr, 0,
                              0, 1);
            RUN_TEST_WITH_ARG(int64_t, int64, xor, flag_d, mype, iter, skip, h_lat, h_size_arr, 0,
                              0, 1);
            RUN_TEST_WITH_ARG(int64_t, int64, xor, flag_d, mype, iter, skip, h_lat, h_size_arr, 0,
                              0, 1);
            RUN_TEST_WITH_ARG(uint32_t, uint32, xor, flag_d, mype, iter, skip, h_lat, h_size_arr, 0,
                              0, 1);
            RUN_TEST_WITH_ARG(unsigned int, uint, xor, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0, 0, 1);
            break;
        }
        case NVSHMEMI_AMO_FETCH_INC: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITHOUT_ARG(unsigned int, uint, fetch_inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(unsigned long, ulong, fetch_inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(unsigned long long, ulonglong, fetch_inc, flag_d, mype, iter, skip,
                                 h_lat, h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(int32_t, int32, fetch_inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(uint32_t, uint32, fetch_inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(uint64_t, uint64, fetch_inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            RUN_TEST_WITHOUT_ARG(int, int, fetch_inc, flag_d, mype, iter, skip, h_lat, h_size_arr,
                                 0);
            RUN_TEST_WITHOUT_ARG(long, long, fetch_inc, flag_d, mype, iter, skip, h_lat, h_size_arr,
                                 0);
            RUN_TEST_WITHOUT_ARG(size_t, size, fetch_inc, flag_d, mype, iter, skip, h_lat,
                                 h_size_arr, 0);
            break;
        }
        case NVSHMEMI_AMO_FETCH_ADD: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned int, uint, fetch_add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(unsigned long, ulong, fetch_add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, fetch_add, flag_d, mype, iter, skip,
                              h_lat, h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(int32_t, int32, fetch_add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(uint32_t, uint32, fetch_add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(uint64_t, uint64, fetch_add, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(int, int, fetch_add, flag_d, mype, iter, skip, h_lat, h_size_arr, 415,
                              0, 0);
            RUN_TEST_WITH_ARG(long, long, fetch_add, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            RUN_TEST_WITH_ARG(size_t, size, fetch_add, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            break;
        }
        case NVSHMEMI_AMO_FETCH_AND: {
            iter = 64;
            skip = 0;
            /* TODO: Figure out a good way to do this with signed types. The bit shifts we do don't
             * mix with signed types. */
            RUN_TEST_WITH_ARG(unsigned long, ulong, fetch_and, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, fetch_and, flag_d, mype, iter, skip,
                              h_lat, h_size_arr, 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF);
            RUN_TEST_WITH_ARG(uint64_t, uint64, fetch_and, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF);
            /* RUN_TEST_WITH_ARG(int64_t, int64, fetch_and, flag_d, mype, iter, skip, h_lat,
             * h_size_arr, 0xFFFFFFFFFFFFFFFF, 0, 0xFFFFFFFFFFFFFFFF); */
            iter = 32;
            /* RUN_TEST_WITH_ARG(int64_t, int64, fetch_and, flag_d, mype, iter, skip, h_lat,
             * h_size_arr, 0xFFFFFFFF, 0, 0xFFFFFFFF); */
            RUN_TEST_WITH_ARG(uint32_t, uint32, fetch_and, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0xFFFFFFFF, 0, 0xFFFFFFFF);
            RUN_TEST_WITH_ARG(unsigned int, uint, fetch_and, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0xFFFFFFFF, 0, 0xFFFFFFFF);
            break;
        }
        case NVSHMEMI_AMO_FETCH_OR: {
            iter = 64;
            skip = 0;
            /* TODO: Figure out a good way to do this with signed types. The bit shifts we do don't
             * mix with signed types. */
            RUN_TEST_WITH_ARG(unsigned long, ulong, fetch_or, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 1, 0xFFFFFFFFFFFFFFFF, 0);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, fetch_or, flag_d, mype, iter, skip,
                              h_lat, h_size_arr, 1, 0xFFFFFFFFFFFFFFFF, 0);
            RUN_TEST_WITH_ARG(uint64_t, uint64, fetch_or, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 1, 0xFFFFFFFFFFFFFFFF, 0);
            /* RUN_TEST_WITH_ARG(int64_t, int64, fetch_or, flag_d, mype, iter, skip, h_lat,
             * h_size_arr, 1, 0xFFFFFFFFFFFFFFFF, 0); */
            iter = 32;
            /* RUN_TEST_WITH_ARG(int64_t, int64, fetch_or, flag_d, mype, iter, skip, h_lat,
             * h_size_arr, 1, 0xFFFFFFFFFFFFFFFF, 0); */
            RUN_TEST_WITH_ARG(uint32_t, uint32, fetch_or, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 1, 0xFFFFFFFF, 0);
            RUN_TEST_WITH_ARG(unsigned int, uint, fetch_or, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 1, 0xFFFFFFFF, 0);
            break;
        }
        case NVSHMEMI_AMO_FETCH_XOR: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned long, ulong, fetch_xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, fetch_xor, flag_d, mype, iter, skip,
                              h_lat, h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(uint64_t, uint64, fetch_xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(int64_t, int64, fetch_xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(int64_t, int64, fetch_xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(uint32_t, uint32, fetch_xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(unsigned int, uint, fetch_xor, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            break;
        }
        case NVSHMEMI_AMO_SWAP: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned int, uint, swap, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0, 0, 1);
            RUN_TEST_WITH_ARG(unsigned long, ulong, swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(int32_t, int32, swap, flag_d, mype, iter, skip, h_lat, h_size_arr, 0,
                              0, 1);
            RUN_TEST_WITH_ARG(uint32_t, uint32, swap, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0, 0, 1);
            RUN_TEST_WITH_ARG(uint64_t, uint64, swap, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              0, 0, 1);
            RUN_TEST_WITH_ARG(int, int, swap, flag_d, mype, iter, skip, h_lat, h_size_arr, 0, 0, 1);
            RUN_TEST_WITH_ARG(long, long, swap, flag_d, mype, iter, skip, h_lat, h_size_arr, 0, 0,
                              1);
            RUN_TEST_WITH_ARG(size_t, size, swap, flag_d, mype, iter, skip, h_lat, h_size_arr, 0, 0,
                              1);
            break;
        }
        case NVSHMEMI_AMO_COMPARE_SWAP: {
            iter = 500;
            skip = 50;
            RUN_TEST_WITH_ARG(unsigned int, uint, compare_swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(unsigned long, ulong, compare_swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(unsigned long long, ulonglong, compare_swap, flag_d, mype, iter, skip,
                              h_lat, h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(int32_t, int32, compare_swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(uint32_t, uint32, compare_swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(uint64_t, uint64, compare_swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            RUN_TEST_WITH_ARG(int, int, compare_swap, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            RUN_TEST_WITH_ARG(long, long, compare_swap, flag_d, mype, iter, skip, h_lat, h_size_arr,
                              415, 0, 0);
            RUN_TEST_WITH_ARG(size_t, size, compare_swap, flag_d, mype, iter, skip, h_lat,
                              h_size_arr, 415, 0, 0);
            break;
        }
        default: {
            fprintf(stderr, "Error, unsupported Atomic op %d.\n", op);
            rc = -1;
            break;
        }
    }

    MAIN_CLEANUP(flag_d, stream, h_tables, 2);
    return rc;
}