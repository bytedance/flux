#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "utils.h"

#define MAX_MSG_SIZE (32 * 1024 * 1024)

#define MAX_ITERS 200
#define MAX_SKIP 20
#define BLOCKS 4
#define THREADS_PER_BLOCK 1024

__global__ void bw(double *data_d, volatile unsigned int *counter_d, int len, int pe, int iter) {
    int i, peer;
    unsigned int counter;
    int tid = (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z);
    int bid = blockIdx.x;
    int nblocks = gridDim.x;

    peer = !pe;
    for (i = 0; i < iter; i++) {
        nvshmemx_double_get_nbi_block(data_d + (bid * (len / nblocks)),
                                      data_d + (bid * (len / nblocks)), len / nblocks, peer);
        // synchronizing across blocks
        __syncthreads();
        if (!tid) {
            __threadfence();
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }
        __syncthreads();
    }

    // synchronizing across blocks
    __syncthreads();
    if (!tid) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            nvshmem_quiet();
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    double *data_d = NULL;
    unsigned int *counter_d;
    int max_blocks = BLOCKS, max_threads = THREADS_PER_BLOCK;
    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_bw = NULL, *h_bw_total = NULL;
    double *d_bw = NULL, *d_bw_sum = NULL;
    bool bidirectional = false;

    int iter = MAX_ITERS;
    int skip = MAX_SKIP;

    float milliseconds;
    hipEvent_t start, stop;

    init_wrapper(&argc, &argv);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    while (1) {
        int c;
        c = getopt(argc, argv, "c:t:hb");

        if (c == -1) break;

        switch (c) {
            case 'c':
                max_blocks = strtol(optarg, NULL, 0);
                break;
            case 't':
                max_threads = strtol(optarg, NULL, 0);
                break;
            case 'b':
                bidirectional = true;
                break;
            default:
            case 'h':
                printf("-c [CTAs] -t [THREADS] -b\n");
                goto finalize;
        }
    }

    data_d = (double *)nvshmem_malloc(MAX_MSG_SIZE);
    CUDA_CHECK(hipMemset(data_d, 0, MAX_MSG_SIZE));

    array_size = floor(std::log2((float)MAX_MSG_SIZE)) + 1;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_bw = (double *)h_tables[1];

    if (bidirectional) {
        h_bw_total = (double *)malloc(sizeof(double) * array_size);

        if (!h_bw_total) {
            fprintf(stderr, "Error: Unable to malloc on the host.\n");
            exit(1);
        }

        memset(h_bw_total, 0, sizeof(double) * array_size);

        /* Allocate on GPU. */
        CUDA_CHECK(hipMalloc((void **)&d_bw, sizeof(double)));
        CUDA_CHECK(hipMalloc((void **)&d_bw_sum, sizeof(double)));
    }

    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

    CUDA_CHECK(hipDeviceSynchronize());

    if (bidirectional || mype == 0) {
        i = 0;
        for (int size = 1024; size <= MAX_MSG_SIZE; size *= 2) {
            h_size_arr[i] = size;
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));
            bw<<<max_blocks, max_threads>>>(data_d, counter_d, size / sizeof(double), mype, skip);
            CUDA_CHECK(hipDeviceSynchronize());
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            hipEventRecord(start);
            bw<<<max_blocks, max_threads>>>(data_d, counter_d, size / sizeof(double), mype, iter);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            hipEventElapsedTime(&milliseconds, start, stop);
            h_bw[i] = size / (milliseconds * (B_TO_GB / (iter * MS_TO_S)));
            nvshmem_barrier_all();

            /* Sum all h_bw of each PE for bidirectional mode. */
            if (bidirectional) {
                CUDA_CHECK(hipMemcpy(d_bw, &h_bw[i], sizeof(double), hipMemcpyHostToDevice));
                nvshmem_double_sum_reduce(NVSHMEM_TEAM_WORLD, d_bw_sum, d_bw, 1);
                CUDA_CHECK(
                    hipMemcpy(&h_bw_total[i], d_bw_sum, sizeof(double), hipMemcpyDeviceToHost));
                CUDA_CHECK(
                    hipMemcpy(&h_bw_total[i], d_bw_sum, sizeof(double), hipMemcpyDeviceToHost));
            }

            i++;
        }
    } else {
        for (int size = 1024; size <= MAX_MSG_SIZE; size *= 2) {
            nvshmem_barrier_all();
        }
    }

    if (mype == 0) {
        double *p_h_bw_tmp = bidirectional ? h_bw_total : h_bw;
        const char *const test_name = bidirectional ? "shmem_get_bw_bidi" : "shmem_get_bw_uni";
        print_table_basic(test_name, "None", "size (Bytes)", "BW", "GB/sec", '+', h_size_arr,
                          p_h_bw_tmp, i);
    }

finalize:

    if (data_d) nvshmem_free(data_d);
    if (h_bw_total) free(h_bw_total);
    if (d_bw) hipFree(d_bw);
    if (d_bw_sum) hipFree(d_bw_sum);
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
