#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define THREADS 512
#define MAX_MSG_SIZE 64 * 1024
#define UNROLL 8

__global__ void pull(int *data_d, int len, int pe, int iter) {
    int i, j, tid, peer;

    peer = !pe;
    tid = threadIdx.x;

    for (i = 0; i < iter; i++) {
        if (!pe) {
            for (j = tid; j < len; j += THREADS) {
                *(data_d + j) = nvshmem_int_g(data_d + j, peer);
            }

            __syncthreads();
        }
    }
}

int main(int c, char *v[]) {
    int mype, npes, size;
    int *data_d = NULL;

    int iter = 200;
    int skip = 20;
    int max_msg_size = MAX_MSG_SIZE;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    float milliseconds;
    hipEvent_t start, stop;

    init_wrapper(&c, &v);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = floor(std::log2((float)max_msg_size)) + 1;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];

    data_d = (int *)nvshmem_malloc(max_msg_size);
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());

    i = 0;
    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        int nelems;
        h_size_arr[i] = size;
        nelems = size / sizeof(int);

        pull<<<1, THREADS>>>(data_d, nelems, mype, skip);
        hipEventRecord(start);
        pull<<<1, THREADS>>>(data_d, nelems, mype, iter);
        hipEventRecord(stop);

        CUDA_CHECK(hipEventSynchronize(stop));

        hipEventElapsedTime(&milliseconds, start, stop);
        /* give latency in us */
        h_lat[i] = (milliseconds * 1000) / iter;
        nvshmem_barrier_all();
        i++;
    }

    if (mype == 0) {
        print_table_v1("shmem_g_latency", "None", "size (Bytes)", "latency", "us", '-', h_size_arr,
                       h_lat, i);
    }

finalize:

    if (data_d) nvshmem_free(data_d);
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
