#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See COPYRIGHT for license information
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <map>
#include <string>
#include <typeinfo>
#include "internal/host/util.h"
#include "internal/non_abi/nvshmemi_h_to_d_coll_defs.cuh"
#include "host/nvshmem_api.h"

#define _THREADS_PER_WARP 32

using namespace std;

map<string, size_t> nvshmemi_fcollect_maxblocksize;

template <typename TYPE>
void nvshmemi_call_fcollect_on_stream_kernel(nvshmem_team_t team, TYPE *dest, const TYPE *source,
                                             size_t nelems, hipStream_t stream) {
    int tmp;
    size_t num_threads_by_elem;
    string type_str(typeid(TYPE).name());
    if (nvshmemi_fcollect_maxblocksize.find(type_str) == nvshmemi_fcollect_maxblocksize.end()) {
        CUDA_RUNTIME_CHECK(hipOccupancyMaxPotentialBlockSize(
            &tmp, (int *)&nvshmemi_fcollect_maxblocksize[type_str],
            fcollect_on_stream_kernel<TYPE>));
    }

    num_threads_by_elem =
        (nelems / _THREADS_PER_WARP + (nelems % _THREADS_PER_WARP ? 1 : 0)) * _THREADS_PER_WARP;

    /* By default select min(occupancy, nelems) */
    int num_threads_per_block = (nvshmemi_fcollect_maxblocksize[type_str] > num_threads_by_elem)
                                    ? num_threads_by_elem
                                    : nvshmemi_fcollect_maxblocksize[type_str];

    /* Use env to override the value */
    if (nvshmemi_options.FCOLLECT_NTHREADS_provided) {
        num_threads_per_block = nvshmemi_options.FCOLLECT_NTHREADS;
    }

    nvshmemi_team_t *teami = nvshmemi_team_pool[team];
    int num_blocks = (nelems >= 65536) ? nvshmemi_options.MAX_CTAS : 1;
    if (num_blocks > 1 && teami->team_dups[1] == NVSHMEM_TEAM_INVALID) {
        NVSHMEMU_FOR_EACH(block_id, num_blocks - 1) {
            nvshmem_team_split_strided(team, 0, 1, nvshmem_team_n_pes(team), NULL, 0,
                                       &(teami->team_dups[block_id + 1]));
            INFO(NVSHMEM_TEAM, "Duplicate team ID: %d of parent team: %d for CTA %zu\n",
                 teami->team_dups[block_id + 1], teami->team_idx, block_id);
        }

        off_t team_dups_offset = offsetof(nvshmemi_team_t, team_dups);
        nvshmemi_team_t *teami_pool_device_addr;
        CUDA_RUNTIME_CHECK(hipMemcpy((void **)&teami_pool_device_addr,
                                      &nvshmemi_device_state.team_pool[team],
                                      sizeof(nvshmemi_team_t *), hipMemcpyDeviceToHost));
        CUDA_RUNTIME_CHECK(hipDeviceSynchronize());
        off_t team_dups_device_addr = (off_t)((char *)teami_pool_device_addr + team_dups_offset);
        CUDA_RUNTIME_CHECK(hipMemcpy((void *)(team_dups_device_addr), &teami->team_dups[0],
                                      sizeof(nvshmem_team_t) * num_blocks, hipMemcpyHostToDevice));
        CUDA_RUNTIME_CHECK(hipDeviceSynchronize());
    }

    fcollect_on_stream_kernel<TYPE>
        <<<num_blocks, num_threads_per_block, 0, stream>>>(team, dest, source, nelems);
    CUDA_RUNTIME_CHECK(hipGetLastError());
}

#define INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(TYPE) \
    template void nvshmemi_call_fcollect_on_stream_kernel<TYPE>(  \
        nvshmem_team_t, TYPE *, const TYPE *, size_t, hipStream_t);
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(uint8_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(uint16_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(uint32_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(uint64_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(int8_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(int16_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(int32_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(int64_t)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(half)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(__hip_bfloat16)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(float)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(char)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(double)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(long long)
INSTANTIATE_NVSHMEMI_CALL_FCOLLECT_ON_STREAM_KERNEL(unsigned long long)
