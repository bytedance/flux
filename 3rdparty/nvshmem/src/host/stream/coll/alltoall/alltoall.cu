#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * See COPYRIGHT for license information
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <map>
#include <string>
#include <typeinfo>

#include "internal/host/util.h"
#include "internal/non_abi/nvshmemi_h_to_d_coll_defs.cuh"

using namespace std;

map<string, size_t> nvshmemi_alltoall_maxblocksize;

template <typename TYPE>
void nvshmemi_call_alltoall_on_stream_kernel(nvshmem_team_t team, TYPE *dest, const TYPE *source,
                                             size_t nelems, hipStream_t stream) {
    int tmp;
    string type_str(typeid(TYPE).name());
    if (nvshmemi_alltoall_maxblocksize.find(type_str) == nvshmemi_alltoall_maxblocksize.end()) {
        CUDA_RUNTIME_CHECK(hipOccupancyMaxPotentialBlockSize(
            &tmp, (int *)&nvshmemi_alltoall_maxblocksize[type_str],
            alltoall_on_stream_kernel<TYPE>));
    }
    int num_threads_per_block = (nvshmemi_alltoall_maxblocksize[type_str] > nelems)
                                    ? nelems
                                    : nvshmemi_alltoall_maxblocksize[type_str];
    int num_blocks = 1;
    alltoall_on_stream_kernel<TYPE>
        <<<num_blocks, num_threads_per_block, 0, stream>>>(team, dest, source, nelems);
    CUDA_RUNTIME_CHECK(hipGetLastError());
}

#define INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(TYPE) \
    template void nvshmemi_call_alltoall_on_stream_kernel<TYPE>(  \
        nvshmem_team_t, TYPE *, const TYPE *, size_t, hipStream_t);
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(uint8_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(uint16_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(uint32_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(uint64_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(int8_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(int16_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(int32_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(int64_t)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(half)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(__hip_bfloat16)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(float)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(char)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(double)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(long long)
INSTANTIATE_NVSHMEMI_CALL_ALLTOALL_ON_STREAM_KERNEL(unsigned long long)
