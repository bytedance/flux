#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include "nvshmem.h"
#include "nvshmemx.h"

#ifdef NVSHMEMTEST_MPI_SUPPORT
#include "mpi.h"
#endif

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

#define THREADS_PER_BLOCK 1024

__global__ void set_and_shift_kernel(float *send_data, float *recv_data, int num_elems, int mype,
                                     int npes) {
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    /* set the corresponding element of send_data */
    if (thread_idx < num_elems) send_data[thread_idx] = mype;

    int peer = (mype + 1) % npes;
    /* Every thread in block 0 calls nvshmemx_float_put_block. Alternatively,
       every thread can call shmem_float_p, but shmem_float_p has a disadvantage
       that when the destination GPU is connected via IB, there will be one rma
       message for every single element which can be detrimental to performance.
       And the disadvantage with shmem_float_put is that when the destination GPU is p2p
       connected, it cannot leverage multiple threads to copy the data to the destination
       GPU. */
    int block_offset = blockIdx.x * blockDim.x;
    nvshmemx_float_put_block(recv_data + block_offset, send_data + block_offset,
                             min(blockDim.x, num_elems - block_offset),
                             peer); /* All threads in a block call the API
                                       with the same arguments */
}

int main(int c, char *v[]) {
    int mype, npes, mype_node;
    float *send_data, *recv_data;
    int num_elems = 8192;
    int num_blocks;

#ifdef NVSHMEMTEST_MPI_SUPPORT
    bool use_mpi = false;
    char *value = getenv("NVSHMEMTEST_USE_MPI_LAUNCHER");
    if (value) use_mpi = atoi(value);
#endif

#ifdef NVSHMEMTEST_MPI_SUPPORT
    if (use_mpi) {
        MPI_Init(&c, &v);
        int rank, nranks;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &nranks);
        MPI_Comm mpi_comm = MPI_COMM_WORLD;

        nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;

        attr.mpi_comm = &mpi_comm;
        nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    } else
        nvshmem_init();
#else
    nvshmem_init();
#endif

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    // application picks the device each PE will use
    CUDA_CHECK(hipSetDevice(mype_node));
    send_data = (float *)nvshmem_malloc(sizeof(float) * num_elems);
    recv_data = (float *)nvshmem_malloc(sizeof(float) * num_elems);
    assert(send_data != NULL && recv_data != NULL);

    assert(num_elems % THREADS_PER_BLOCK == 0); /* for simplicity */
    num_blocks = num_elems / THREADS_PER_BLOCK;

    set_and_shift_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(send_data, recv_data, num_elems, mype,
                                                            npes);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    /* Do data validation */
    float *host = new float[num_elems];
    CUDA_CHECK(hipMemcpy(host, recv_data, num_elems * sizeof(float), hipMemcpyDefault));
    int ref = (mype - 1 + npes) % npes;
    bool success = true;
    for (int i = 0; i < num_elems; ++i) {
        if (host[i] != ref) {
            printf("Error at %d of rank %d: %f\n", i, mype, host[i]);
            success = false;
            break;
        }
    }

    if (success) {
        printf("[%d of %d] run complete \n", mype, npes);
    } else {
        printf("[%d of %d] run failure \n", mype, npes);
    }

    nvshmem_free(send_data);
    nvshmem_free(recv_data);

    nvshmem_finalize();

#ifdef NVSHMEMTEST_MPI_SUPPORT
    if (use_mpi) MPI_Finalize();
#endif
    return 0;
}
