#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

#define MPI_CHECK(stmt)                                                                         \
    do {                                                                                        \
        int result = (stmt);                                                                    \
        if (MPI_SUCCESS != result) {                                                            \
            fprintf(stderr, "[%s:%d] MPI failed with error %d \n", __FILE__, __LINE__, result); \
            exit(-1);                                                                           \
        }                                                                                       \
    } while (0)

__global__ void simple_shift(int *target, int mype, int npes) {
    int peer = (mype + 1) % npes;
    nvshmem_int_p(target, mype, peer);
}

int main(int c, char *v[]) {
    int *target;
    int rank, nranks;
    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;
    int mype, npes, mype_node;

    MPI_CHECK(MPI_Init(&c, &v));
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nranks));

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    // application picks the device each PE will use
    CUDA_CHECK(hipSetDevice(mype_node));
    target = (int *)nvshmem_malloc(sizeof(int));

    simple_shift<<<1, 1>>>(target, mype, npes);
    CUDA_CHECK(hipDeviceSynchronize());

    printf("[%d of %d] run complete \n", mype, npes);

    nvshmem_free(target);

    nvshmem_finalize();
    MPI_CHECK(MPI_Finalize());
    return 0;
}
